#include "hip/hip_runtime.h"
nclude <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "ANNCUDA.h"

#define CEIL(x,y) (((x)+(y)-1) / (y))
hipblasHandle_t handle;				/*  */


void StartCUDA(void) {
    hipblasStatus_t status;
  	/* initiate CUBLAS */
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf(" Failed to initialise CUBLAS");
        exit(0);
    }    
}
void StopCUDA(void) {
	/* destroy the context on the GPU */
    hipblasDestroy(handle);
}

//-------------------------------------------------------------------------------------------------------------------
/* routines transfer of data from host(device) to device(host)*/
//-------------------------------------------------------------------------------------------------------------------
void SyncDev2Host(void *devPtr, void *hostPtr, size_t size) {
    hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
}

void initialiseDeviceArrayWithZero(void *devPtr,size_t size){
	hipMemset(devPtr, 0, size);
}
/*  */
void SyncHost2Dev(void *hostPtr, float *devPtr, size_t size) {
    hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);	
}

/*  */
void DevDispose(void *devPtr) {
    hipFree(devPtr);
}

/*  */
void DevNew(void **devAddr, size_t size) {
    hipMalloc(devAddr, size);
}

//-------------------------------------------------------------------------------------------------------------------
/* this section of the code presents code contains auxillary  functions that are frequently used*/
//-------------------------------------------------------------------------------------------------------------------
void AddNSegmentCUDA(float * srcPtr, int segLen, float * dstPtr, float  lambda) {
	hipblasStatus_t status;
	status = hipblasSaxpy(handle, segLen, &lambda, srcPtr, 1, dstPtr, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("AddNSegmentCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }
}

/*  */
void ScaleNSegmentCUDA(int segLen, float scale, float * valPtr) {
    hipblasStatus_t status;
	status = hipblasSscal(handle, segLen, &scale, valPtr, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ScaleNSegmentCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }
}

void SubNSegmentCUDA(float *srcPtr, int segLen, float *dstPtr,float lambda) {
    hipblasStatus_t status;
    status = hipblasSaxpy(handle, segLen, &lambda, srcPtr, 1, dstPtr, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("SubNSegmentCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }
}

void CopyMatrixOrVecCUDA(float * src , float *dest, int dim){
    const float * sr = src;
	hipblasStatus_t status;
    status = hipblasScopy(handle,dim, sr, 1,dest, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CopyMatrixOrVecCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }

}

void computeDotProductCUDA(float * vectorL, float * vectorR,int dim,float  *result){
	hipblasStatus_t status;
    status= hipblasSdot (handle, dim, vectorL, 1, vectorR, 1, result);
     if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("computeDotProductCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }
}
//-------------------------------------------------------------------------------------------------------------------
/*The following routines are used for forward propagation*/
//-------------------------------------------------------------------------------------------------------------------

void CopyNSegmentCUDA(float *srcPtr, int segLen, float *dstPtr) {
    hipblasStatus_t status;
    status = hipblasScopy(handle, segLen, srcPtr, 1, dstPtr, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CopyNSegmentCUDA: CUBLAS library copy function failed\n");
        exit(0);
    }
}

void HNBlasTNgemmCUDA(int m, int n, int k, float alpha, float *A, float *B, float beta, float *C) {
    hipblasStatus_t status;
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, C, m);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("HNBlasTNgemmCUDA: CUBLAS library gemm function failed\n");
        exit(0);
    }
}

__global__ void HKern_ApplySigmoidAct(float *srcPtr, int len, float *dstPtr) {
    int pos;
    float floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = -1.0 * srcPtr[pos];
        dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
    }
}
void ApplySigmoidActCUDA(float *srcPtr, int len, float *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){
        printf("ApplySigmoidActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }
    HKern_ApplySigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

__global__ void HKern_ApplyTanHAct(float *srcPtr, int len, float *dstPtr) {
    int pos;
    float floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        floatVal = exp(floatVal);
        dstPtr[pos] = (floatVal - 1 / floatVal) / (floatVal + 1 / floatVal);
    }
}
void ApplyTanHActCUDA(float *srcPtr, int len, float *dstPtr) {
    int nBlocks;
    
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){
        printf( "ApplyTanHActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }	
    HKern_ApplyTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

__global__ void HKern_ApplySoftmaxAct(float *srcPtr, int row, int col, float *dstPtr) {
    int frame, i, base, off;
    float den, floatVal;

    frame = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (frame < row) {
        den = 0.0;
        base = frame * col;
        for (i = 0, off = base; i < col; ++i, ++off) {
            floatVal = srcPtr[off];
            floatVal = exp(floatVal);
            dstPtr[off] = floatVal;
            den += floatVal;
        }
        for (i = 0, off = base; i < col; ++i, ++off) {
            dstPtr[off] /= den;
        }
    }
}
void ApplySoftmaxActCUDA(float *srcPtr, int row, int col, float *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){}
        printf("ApplySoftmaxActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }
    HKern_ApplySoftmaxAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, dstPtr);
}

//-------------------------------------------------------------------------------------------------------------------
/*The following routines are used for back propagation*/
//-------------------------------------------------------------------------------------------------------------------
__global__ void  HKern_fillArrayWithValue(float* array,float value,int len){
	int pos;
	pos = (blockIdx.x * blockDim.x) + threadIdx.x
	if (pos<len){
		array[pos] = value;
	}
}
void sumColsOfMatrix(float *dyFeatMat,float *dbFeatMat,int dim,int batchsamples){
 	int nBlocks;
 	float * array;
 	const float alpha = 1;
 	const float beta = 0;
 	hipMalloc((**void)&array,sizeof(float)*batchsamples);
	nBlocks = CEIL(batchsamples, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){}
        printf("ApplySoftmaxActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }
    HKern_fillArrayWithValue<<<nBlocks, THREADPERBLOCK>>>(array,value,batchsamples);
    
    hipblasSgemv(handle,HIPBLAS_OP_N, dim,batchsamples,&alpha,dyFeatMat,dim,ones,1,&beta,dbFeatMat,1);
	DevDispose(array);
}


__global__ void HKern_ApplyDerivativeSigmoidAct(float * srcPtr, int len, float *dstPtr){
	int pos;
    float floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = dstPtr[pos] * (srcPtr[pos] * (1 - srcPtr[pos])) ;
    }
}

void  computeSigmoidDrvCUDA(float *srcPtr, int len, float *dstPtr){
	 int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){
        printf("ApplySigmoidActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }
    HKern_ApplyDerivativeSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

__global__ void HKern_ApplyDerivativeTanHAct(float * srcPtr, int len, float *dstPtr){
	int pos;
    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = dstPtr[pos] * (1 - (srcPtr[pos] * srcPtr[pos]));
    }
}

void  computeTanHDrvCUDA(float *srcPtr, int len, float *dstPtr){
	int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){
        printf("ApplySigmoidActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }
    HKern_ApplyDerivativeTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

void HNBlasNNgemmCUDA(int m, int n, int k, float alpha, float *A, float *B, float beta, float *C) {
    hipblasStatus_t status;
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("HNBlasNNgemmCUDA: CUBLAS library gemm function failed\n");
        exit(0);
    }
}
void HNBlasNTgemmCUDA(int m, int n, int k, float alpha, float *A, float *B, float beta, float *C) {
    hipblasStatus_t status;
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("HNBlasNNgemmCUDA: CUBLAS library gemm function failed\n");
        exit(0);
    }

}
/-------------------------------------------------------------------------------------------------------------------
/*The following routines are used for computing the hessian of the loss function with respect to network outputs*/
//-------------------------------------------------------------------------------------------------------------------
__global__ void HKern_AddElementstoDiagonalOfMatrix(float * lhs , float * rhs , int dim float * dst){
	int pos;
	pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos<len){
    	dst[pos*(dim+1)] = lhs[pos*(dim+1)]+ rhs[pos]; 
    }
}

void AddElementstoDiagonalOfMatrix(float * lhs , float * rhs , int dim, float * dst){
	int nBlocks;
	nBlocks = CEIL(dim, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM){
        printf("ApplySigmoidActCUDA: Block number exceeds the maximum\n");
    	exit(0);
    }

}



